#include "hip/hip_runtime.h"
/*
Try accelerating splotch with CUDA. July 2009.
Copyright things go here.
*/

//#include "splotch_kernel.h"
#include "splotch_cuda.h"

//MACROs
#define Pi 3.14159265358979323846264338327950288
#define get_xy_from_sn(sn, xmin, ymin, ymax, x, y)\
        {int x1 =sn/(ymax-ymin); int y1 =sn-x1*(ymax-ymin);\
         x  =x1 +xmin; y  =y1 +ymin;}
#define get_sn_from_xy(x,y,maxy,miny, sn)\
    {sn =x*(maxy-miny) +y;}

#define get_minmax(minv, maxv, val) \
         minv=min(minv,val); \
         maxv=max(maxv,val);
#define MAXSIZE 1000

/////////constant memory declaration /////////////////////
__constant__ cu_color_map_entry dmap[MAXSIZE];
__constant__ int ptype_points[10];
__constant__ cu_param dparams;

/////////help functions///////////////////////////////////
__device__ float    my_asinh(float val)
  {
  return log(val+sqrt(1.+val*val));
  }

__device__ void my_normalize(float minv, float maxv, float &val)
  {
  if (minv!=maxv) val =  (val-minv)/(maxv-minv);
  }

__device__ void clamp (float minv, float maxv, float &val)
  {
  val = min(maxv, max(minv, val));
  }

//fetch a color from color table on device
__device__ cu_color get_color(int ptype, float val, int mapSize, int ptypes)
  {
  __shared__ int map_size;
  __shared__ int map_ptypes;

  map_size = mapSize;
  map_ptypes = ptypes;
  //first find the right entry for this ptype
  int     start, end;
  start =ptype_points[ptype];
  if ( ptype == map_ptypes-1)//the last type
    end =map_size-1;
  else
    end =ptype_points[ptype+1]-1;

  //search the section of this type to find the val
  int i=start;
  while ((val>dmap[i+1].val) && (i<end)) ++i;

  const float fract = (val-dmap[i].val)/(dmap[i+1].val-dmap[i].val);
  cu_color clr1=dmap[i].color, clr2=dmap[i+1].color;
  cu_color        clr;
  clr.r =clr1.r + fract*(clr2.r-clr1.r);
  clr.g =clr1.g + fract*(clr2.g-clr1.g);
  clr.b =clr1.b + fract*(clr2.b-clr1.b);

  return clr;
  }

__global__ void k_post_process(cu_color *pic, int n)
  {
  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >=n)
    m =n;

  //each pic[m] should do the same calc, so sequence does not matter!
  pic[m].r =1.0 - exp( pic[m].r);
  pic[m].g =1.0 - exp( pic[m].g);
  pic[m].b =1.0 - exp( pic[m].b);
  }

__global__ void k_combine
  (int minx, int miny, int maxx, int maxy, int xres, int yres,
  cu_particle_splotch *p, int pStart, int pEnd, cu_fragment_AeqE *fbuf, cu_color *pic)
  {
  int m =blockIdx.x *blockDim.x + threadIdx.x;
  int n =(maxx-minx)*(maxy-miny);
  if (m >=n)
    m =n;

  //get global coordinate point(x,y) of this thread
  int point_x, point_y;
  get_xy_from_sn(m, minx, miny, maxy, point_x, point_y);

  //go through all particles, for each particle p if point(x,y) is in its region
  //p(minx,miny, maxx,maxy) do the following.
  //find the sequencial number sn1 in p(minx,miny, maxx,maxy), the fragment we are looking
  //for in fragment buffer is fragBuf[ sn1+p.posInFBuf ]
  //grab the fragment f(deltaR,deltaG,deltaB)
  //find the sequencial number sn2 of point(x,y) in the output pic.
  //pic[sn2] += f
  int sn1, sn2, local_x, local_y, fpos;
  for (int i=pStart; i<=pEnd; i++)
    {
    if ( point_x >=p[i].minx && point_x<p[i].maxx &&
         point_y >=p[i].miny && point_y<p[i].maxy)
      {
      local_x =point_x -p[i].minx;
      local_y =point_y -p[i].miny;
      get_sn_from_xy(local_x, local_y, p[i].maxy, p[i].miny,sn1);
      fpos =sn1 +p[i].posInFragBuf;

      get_sn_from_xy(point_x, point_y, yres,0, sn2);
      pic[sn2].r +=fbuf[fpos].aR;
      pic[sn2].g +=fbuf[fpos].aG;
      pic[sn2].b +=fbuf[fpos].aB;
      }
    }
  }

//device render function k_render1
__global__ void k_render1
  (cu_particle_splotch *p, int nP,
  void *buf, bool a_eq_e, float grayabsorb, int mapSize, int types)
  {
  //first get the index m of this thread
  int m;

  m =blockIdx.x *blockDim.x + threadIdx.x;
  if (m >=nP)//m goes from 0 to nP-1
    return;

  // coloring
  int ptype = p[m].type;
  float col1=p[m].e.r,col2=p[m].e.g,col3=p[m].e.b;
  clamp (0.0000001,0.9999999,col1);
  if (dparams.col_vector[ptype])
    {
    clamp (0.0000001,0.9999999,col2);
    clamp (0.0000001,0.9999999,col3);
    }
  float intensity=p[m].I;
  clamp (0.0000001,0.9999999,intensity);
  intensity *= dparams.brightness[ptype];

  cu_color e;
  if (dparams.col_vector[ptype])   // color from file
    {
    e.r=col1*intensity;
    e.g=col2*intensity;
    e.b=col3*intensity;
    }
  else   // get color, associated from physical quantity contained in e.r, from lookup table
    {
  //first find the right entry for this ptype
      if (ptype<types)
      {
        e = get_color(ptype, col1, mapSize, types);
        e.r *= intensity;
        e.g *= intensity;
        e.b *= intensity;
      }
      else
      { e.r =e.g =e.b =0.0; }
    }

  //make fbuf the right type
  cu_fragment_AeqE        *fbuf;
  cu_fragment_AneqE       *fbuf1;
  if (a_eq_e)
    fbuf =(cu_fragment_AeqE*) buf;
  else
    fbuf1 =(cu_fragment_AneqE*)buf;

  //now do the rendering
  const float powtmp = pow(Pi,1./3.);
  const float sigma0 = powtmp/sqrt(2*Pi);

  const float r = p[m].r;
  const float radsq = 2.25*r*r;
  const float stp = -0.5/(r*r*sigma0*sigma0);

  cu_color q; //e=p[m].e;
  if (!a_eq_e)
   {
     q.r = e.r/(e.r+grayabsorb);
     q.g = e.g/(e.g+grayabsorb);
     q.b = e.b/(e.b+grayabsorb);
   }
  const float intens = -0.5/(2*sqrt(Pi)*powtmp);
  e.r*=intens; e.g*=intens; e.b*=intens;

  const float posx=p[m].x, posy=p[m].y;
  unsigned int fpos =p[m].posInFragBuf;

  if (a_eq_e)
  {
    for (int x=p[m].minx; x<p[m].maxx; ++x)
    {
     float dxsq=(x-posx)*(x-posx);
     for (int y=p[m].miny; y<p[m].maxy; ++y)
      {
        float dsq = (y-posy)*(y-posy) + dxsq;
        if (dsq<radsq)
        {
          float att = __expf(stp*dsq);
          fbuf[fpos].aR = att*e.r;
          fbuf[fpos].aG = att*e.g;
          fbuf[fpos].aB = att*e.b;
        }
        else
        {
          fbuf[fpos].aR =0.0;
          fbuf[fpos].aG =0.0;
          fbuf[fpos].aB =0.0;
        }
      //for each (x,y)
      fpos++;
      }//y
    }//x
  }
  else
  {
    for (int x=p[m].minx; x<p[m].maxx; ++x)
    {
     float dxsq=(x-posx)*(x-posx);
     for (int y=p[m].miny; y<p[m].maxy; ++y)
      {
        float dsq = (y-posy)*(y-posy) + dxsq;
        if (dsq<radsq)
        {
          float att = __expf(stp*dsq);
          float   expm1;
          expm1 =__expf(att*e.r)-1.0;
          fbuf1[fpos].aR = expm1;
          fbuf1[fpos].qR = q.r;
          expm1 =__expf(att*e.g)-1.0;
          fbuf1[fpos].aG = expm1;
          fbuf1[fpos].qG = q.g;
          expm1 =__expf(att*e.b)-1.0;
          fbuf1[fpos].aB = expm1;
          fbuf1[fpos].qB = q.b;
        }
        else
        {
          fbuf1[fpos].aR =0.0;
          fbuf1[fpos].aG =0.0;
          fbuf1[fpos].aB =0.0;
          fbuf1[fpos].qR =1.0;
          fbuf1[fpos].qG =1.0;
          fbuf1[fpos].qB =1.0;
        }
      //for each (x,y)
      fpos++;
      }//y
    }//x
  }
 }


//Transform by kernel
__global__ void k_transform
  (cu_particle_sim *p, cu_particle_splotch *p2, int n)
  {

  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >n) m =n;

  //now do x,y,z
  float x,y,z;
  x =p[m].x*dparams.p[0] + p[m].y*dparams.p[1] + p[m].z*dparams.p[2] + dparams.p[3];
  y =p[m].x*dparams.p[4] + p[m].y*dparams.p[5] + p[m].z*dparams.p[6] + dparams.p[7];
  z =p[m].x*dparams.p[8] + p[m].y*dparams.p[9] + p[m].z*dparams.p[10]+ dparams.p[11];

  //do r
  float xfac = dparams.xfac;
  const float   res2 = 0.5*dparams.xres;
  const float   ycorr = .5f*(dparams.yres-dparams.xres);
  if (!dparams.projection)
    {
    x = res2 * (x+dparams.fovfct*dparams.dist)*xfac;
    y = res2 * (y+dparams.fovfct*dparams.dist)*xfac + ycorr;
    }
  else
    {
    xfac=1./(dparams.fovfct*z);
    x = res2 * (x+dparams.fovfct*z)*xfac;
    y = res2 * (y+dparams.fovfct*z)*xfac + ycorr;
    }

  float r = p[m].r;
  p[m].I /= r;
  r *= res2*xfac;

  const float rfac= sqrt(r*r + 0.25*dparams.minrad_pix*dparams.minrad_pix)/r;
  r *= rfac;
  p2[m].I = p[m].I/rfac;

  p2[m].isValid = false;

  // compute region occupied by the partile
  const float rfacr=dparams.rfac*r;
  int minx=int(x-rfacr+1);
  if (minx>=dparams.xres) return;
  minx=max(minx,0);

  int maxx=int(x+rfacr+1);
  if (maxx<=0) return;
  maxx=min(maxx,dparams.xres);
  if (minx>=maxx) return;

  int miny=int(y-rfacr+1);
  if (miny>=dparams.yres) return;
  miny=max(miny,0);

  int maxy=int(y+rfacr+1);
  if (maxy<=0) return;
  maxy=min(maxy,dparams.yres);
  if (miny>=maxy) return;

  p2[m].minx =minx;  p2[m].miny =miny;
  p2[m].maxx =maxx;  p2[m].maxy =maxy;

  p2[m].isValid = true;
  p2[m].x = x;
  p2[m].y = y;
  p2[m].r = r;
  p2[m].e.r = (float) p[m].e.r;
  p2[m].e.g = (float) p[m].e.g;
  p2[m].e.b = (float) p[m].e.b;
  p2[m].type = p[m].type;

  }


//colorize by kernel
__global__ void k_colorize
  (int n, cu_particle_splotch *p2, int mapSize, int types)
  {

  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >n) m =n;
 
  int ptype = p2[m].type;
  float col1=p2[m].e.r,col2=p2[m].e.g,col3=p2[m].e.b;
  clamp (0.0000001,0.9999999,col1);
  if (dparams.col_vector[ptype])
    {
    clamp (0.0000001,0.9999999,col2);
    clamp (0.0000001,0.9999999,col3);
    }
  float intensity=p2[m].I;
  clamp (0.0000001,0.9999999,intensity);
  intensity *= dparams.brightness[ptype];

  cu_color e;
  if (dparams.col_vector[ptype])   // color from file
    {
    e.r=col1*intensity;
    e.g=col2*intensity;
    e.b=col3*intensity;
    }
  else   // get color, associated from physical quantity contained in e.r, from lookup table
    {
  //first find the right entry for this ptype
      if (ptype<types)
      {
        e = get_color(ptype, col1, mapSize, types);
        e.r *= intensity;
        e.g *= intensity;
        e.b *= intensity;
      }
      else
      { e.r =e.g =e.b =0.0; }
    }
  p2[m].e=e;

  }



