#include "hip/hip_runtime.h"
#ifndef __KERNEL__
#define __KERNEL__

#include "cuda/splotch_cuda.h"
#include <cstdio>

//#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
//#define printf(f, ...) ((void)(f, __VA_ARGS__),0)
//#endif


//MACROs
#define Pi 3.141592653589793238462643383279502884197
#define MAXSIZE 1000

/////////constant memory declaration /////////////////////

__constant__ cu_color_map_entry dmap[MAXSIZE];
__constant__ int ptype_points[10];
__constant__ cu_param dparams;

//help functions

__device__ __forceinline__ void clamp (float minv, float maxv, float &val)
  {
  val = min(maxv, max(minv, val));
  }

__device__ __forceinline__   double my_asinh (double val)
  { return log(val+sqrt(1.+val*val)); }

//fetch a color from color table on device
__device__ __forceinline__ cu_color get_color(int ptype, float val, int map_size, int map_ptypes)
  {
  //first find the right entry for this ptype
  int     start, end;
  start = ptype_points[ptype];
  if ( ptype == map_ptypes-1)//the last type
    end = map_size-1;
  else
    end = ptype_points[ptype+1]-1;

  //search the section of this type to find the val
  int i=start;
  while ((val>dmap[i+1].val) && (i<end)) ++i;

  const float fract = (val-dmap[i].val)/(dmap[i+1].val-dmap[i].val);
  cu_color clr1=dmap[i].color, clr2=dmap[i+1].color;
  cu_color        clr;
  clr.r =clr1.r + fract*(clr2.r-clr1.r);
  clr.g =clr1.g + fract*(clr2.g-clr1.g);
  clr.b =clr1.b + fract*(clr2.b-clr1.b);

  return clr;
  }

//Transform+coloring by kernel
__global__ void k_process
  (cu_particle_sim *p, int *p_active, int n, int mapSize, int types, int tile_sidex, int tile_sidey, int width, int nxtiles, int nytiles)
  {
  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >=n) return;


  int ptype = p[m].type;
  float r = p[m].r;
  float er = p[m].e.r;
  float eg = p[m].e.g;
  float eb = p[m].e.b;
  float I = p[m].I;

    // Normalization and clamping 

#ifndef NO_I_NORM
  // Norm and clamp I
    if (dparams.inorm_maxs[ptype]==dparams.inorm_mins[ptype])
      I = 1;
    else
      I = (max(dparams.inorm_mins[ptype],min(dparams.inorm_maxs[ptype],I))-dparams.inorm_mins[ptype])/(dparams.inorm_maxs[ptype]-dparams.inorm_mins[ptype]);
#endif

  // Norm and clamp er
    if (dparams.cnorm_maxs[ptype]==dparams.cnorm_mins[ptype])
      er = 1;
    else
      er = (max(dparams.cnorm_mins[ptype],min(dparams.cnorm_maxs[ptype],er))-dparams.cnorm_mins[ptype])/(dparams.cnorm_maxs[ptype]-dparams.cnorm_mins[ptype]);
  
  // If col_vector[t]
  // norm and clamp eg and eb
    if(dparams.col_vector[ptype])
    {
      if (dparams.cnorm_maxs[ptype]==dparams.cnorm_mins[ptype])
        eg = 1;
      else
        eg = (max(dparams.cnorm_mins[ptype],min(dparams.cnorm_maxs[ptype],er))-dparams.cnorm_mins[ptype])/(dparams.cnorm_maxs[ptype]-dparams.cnorm_mins[ptype]);

      if (dparams.cnorm_maxs[ptype]==dparams.cnorm_mins[ptype])
        eb = 1;
      else
        eb = (max(dparams.cnorm_mins[ptype],min(dparams.cnorm_maxs[ptype],er))-dparams.cnorm_mins[ptype])/(dparams.cnorm_maxs[ptype]-dparams.cnorm_mins[ptype]);
    }

 // cu_color e;
 // e.r=p[m].e.r;
 // e.g=p[m].e.g;
 // e.b=p[m].e.b;

  //now do x,y,z
 // float zminval = 0.0;
 // float zmaxval = 1e23;
  float x,y,z;
  x =p[m].x*dparams.p[0] + p[m].y*dparams.p[1] + p[m].z*dparams.p[2] + dparams.p[3];
  y =p[m].x*dparams.p[4] + p[m].y*dparams.p[5] + p[m].z*dparams.p[6] + dparams.p[7];
  z =p[m].x*dparams.p[8] + p[m].y*dparams.p[9] + p[m].z*dparams.p[10]+ dparams.p[11];

  if(z <= 0.0f){p[m].active = false; p_active[m]=-1;return;};
  if(z >= 1e23){p[m].active = false; p_active[m]=-1;return;};
  //do r
  float xfac2 = dparams.xfac;
  //const float   res2 = 0.5f*dparams.xres;
  //const float   ycorr = 0.5f*(dparams.yres-dparams.xres);
  if (!dparams.projection)
    {
    x = 0.5f*dparams.xres * (x+dparams.fovfct*dparams.dist)*xfac2;
    y = 0.5f*dparams.xres * (y+dparams.fovfct*dparams.dist)*xfac2 + 0.5f*(dparams.yres-dparams.xres);
    }
  else
    {
    xfac2=1.f/(dparams.fovfct*z);
    x = 0.5f*dparams.xres * (x+dparams.fovfct*z)*xfac2;
    y = 0.5f*dparams.xres * (y+dparams.fovfct*z)*xfac2 +  0.5f*(dparams.yres-dparams.xres);
    }

#ifdef SPLOTCH_CLASSIC
  I *= 0.5f*dparams.bfak/r;
  r*= sqrtf(2.f)*dparams.sigma0/dparams.h2sigma;  
#else
  //I *= 8.f/(Pi*r*r*r);  //SPH kernel normalization
  //I *= dparams.h2sigma*sqrtf(Pi)*r;  //integral through the center
  I *= 8.f*dparams.h2sigma/(sqrtf(Pi)*r*r);
#endif

  r *= 0.5f*dparams.xres*xfac2;
  const float rcorr= sqrtf(r*r + dparams.minrad_pix*dparams.minrad_pix)/r;
  r *= rcorr;
#ifdef SPLOTCH_CLASSIC
  I /= rcorr;
#else
  I /= rcorr*rcorr;
#endif
  I *= dparams.brightness[ptype];

  p[m].active = false;
  p_active[m] = -1;	// non active particle

  // compute region occupied by the partile
  //float raux=dparams.rfac;
  const float rfacr=dparams.rfac*r;
  int minx=int(x-rfacr+1.f);
  if (minx>=dparams.xres) return;
  minx=max(minx,0);

  int maxx=int(x+rfacr+1.f);
  if (maxx<=0) return;
  maxx=min(maxx,dparams.xres);
  if (minx>=maxx) return;

  int miny=int(y-rfacr+1.f);
  if (miny>=dparams.yres) return;
  miny=max(miny,0);

  int maxy=int(y+rfacr+1.f);
  if (maxy<=0) return;
  maxy=min(maxy,dparams.yres);
  if (miny>=maxy) return;
 
  p[m].active = true;
  
  p[m].x = x;
  p[m].y = y;
  p[m].r = r;
  p[m].I = I;

//coloring
// get color, associated from physical quantity contained in e.r, from lookup table
  cu_color e;
  e.r=er;
  e.g=eg;
  e.b=eb;

  if (!dparams.col_vector[ptype])
     e = get_color(ptype, e.r, mapSize, types);

  p[m].e.r = e.r*I;
  p[m].e.g = e.g*I;
  p[m].e.b = e.b*I; 
  
  // manage particles outside the image but that influence it
  if(x < 0.0 || x >= (float)dparams.xres){p_active[m] = -2; return;};
  if(y < 0.0 || y >= (float)dparams.yres){p_active[m] = -2; return;};
  // active particle = tile_id to which it belongs to
  /////////////CLAAAAA p_active[m] = int(y)/tile_sidey + int(x)/tile_sidex*nytiles; 
  p_active[m] = int(y/float(tile_sidey)) + int(x/float(tile_sidex))*nytiles; 
  //if (p_active[m] < 0 || p_active[m] > nxtiles*nytiles) {printf("x=%f, y=%f, flag=%d\n",x,y,p_active[m]);}
  if ((maxx-minx)*(maxy-miny) <= 1) p_active[m] = nxtiles*nytiles; // point-like particles 
  if (int(rfacr)>=width) 
  {
      p_active[m] = -2; // particle to be removed and copied back to the host 
      //printf("x=%f, y=%f, rfacr=%d, WIDTH=%d \n",p[m].r,raux,int(rfacr),width);
  }
}
//---------------------------------------------------------------------------------
// Ranging - Tim Dykes
// Calculates logs, asinh is commented out because if it is used
// it is done on host
__global__ void k_range(int nP, cu_particle_sim *p)
{

  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >=nP) return;

  // Get current particle type
  int ptype = p[m].type;

  // Check if we need to log10 intensity
 // 
  if (dparams.log_int[ptype])
  { 
    if(p[m].I > 0)
        p[m].I = log10(p[m].I);
    else
        p[m].I = -38;
  }

  if (dparams.log_col[ptype])
  {
    if(p[m].e.r > 0)
      {
      p[m].e.r = log10(p[m].e.r);
      }
    else
      p[m].e.r =-38;
  }
//else
//{
//  if (dparams.asinh_col[ptype])
//    p[m].e.r = my_asinh(p[m].e.r);
//}

  if (dparams.col_vector[ptype])
  {
    if (dparams.log_col[ptype])
    {
      p[m].e.g = log10(p[m].e.g);
      p[m].e.b = log10(p[m].e.b);
    }
//  if (dparams.asinh_col[ptype])
//  {
//    p[m].e.g = my_asinh(p[m].e.g);
//    p[m].e.b = my_asinh(p[m].e.b);
//  }
  }

}
//---------------------------------------------------------------------------------

//colorize by kernel
/*__global__ void k_colorize
  (cu_particle_sim *p2, int mapSize, int types, int n)
  {
  //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >= n) return; 

  int ptype = p2[m].type;
  cu_color e;  
    e.r=p2[m].e.r;
    e.g=p2[m].e.g;
    e.b=p2[m].e.b;

  float intensity = p2[m].I;
  intensity *= dparams.brightness[ptype];

// get color, associated from physical quantity contained in e.r, from lookup table
  if (!dparams.col_vector[ptype])
     e = get_color(ptype, e.r, mapSize, types);

  p2[m].e.r = e.r*intensity;;
  p2[m].e.g = e.g*intensity;;
  p2[m].e.b = e.b*intensity;;
 }
*/


// change of linear coordinate: from tile to global image
// lpix -> (x,y) -> (X,Y) -> gpix 
__device__ int pixelLocalToGlobal(int lpix, int xo, int yo, int width, int tile_sidey)
{
  // global 2D coordinates
  int x = xo + lpix/(tile_sidey+2*width);
  int y = yo + lpix%(tile_sidey+2*width);

  return x*dparams.yres+y;
}

#define NPSIZE 64

//device render function k_render1
// a_eq_e = false is not supported
__global__ void k_render1
  (int nP, cu_particle_sim *part, int *tileId, int *tilepart, cu_color *pic, cu_color *pic1, cu_color *pic2, cu_color *pic3, int tile_sidex, int tile_sidey, int width, int nytiles)
{
   extern __shared__ cu_color Btile[];
   __shared__ int local_chunk_length, end;
   __shared__ cu_color e[NPSIZE];
   __shared__ float radsq[NPSIZE], stp[NPSIZE];
   __shared__ float posx[NPSIZE], posy[NPSIZE];
   __shared__ int minx[NPSIZE], maxx[NPSIZE], miny[NPSIZE], maxy[NPSIZE];
   
   int tileBsize = (tile_sidex+2*width)*(tile_sidey+2*width);
   int tile = tileId[blockIdx.x];	// tile number 

   if (threadIdx.x == 0)
   {
      end = tilepart[blockIdx.x];
      if (blockIdx.x == 0) local_chunk_length = end;
      else local_chunk_length = end - tilepart[blockIdx.x-1];
      end--;
   }
   __syncthreads();

   int xo = (tile/nytiles)*tile_sidex - width;  // Btile origin x
   int yo = (tile%nytiles)*tile_sidey - width;  // Btile origin y

  //inizialise Btile
  for (int i=threadIdx.x; i<tileBsize; i=i+blockDim.x) 
  {
     Btile[i].r = 0.0f;  Btile[i].g = 0.0f;   Btile[i].b = 0.0f;
  }
   __syncthreads();

  int x,y,k;
  int j = 0;
  int last = min(NPSIZE, blockDim.x);
  //now do the rendering: each thread processes a pixel of particle i

  while (j < local_chunk_length) 
  {
      k = threadIdx.x; 
      if(k < last)
      {
        cu_particle_sim p = part[end-k-j];
        e[k] = p.e;
        posx[k] = p.x; posy[k] = p.y;
        float rfacr = dparams.rfac*p.r;
        radsq[k] = rfacr*rfacr;
        stp[k] = -1.f/(dparams.h2sigma*dparams.h2sigma*p.r*p.r);

        minx[k]=int(p.x-rfacr+1.f);
        minx[k]=max(minx[k],0);
        maxx[k]=int(p.x+rfacr+1.f);
        maxx[k]=min(maxx[k],dparams.xres); 
        miny[k]=int(p.y-rfacr+1.f);
        miny[k]=max(miny[k],0);
        maxy[k]=int(p.y+rfacr+1.f);
        maxy[k]=min(maxy[k],dparams.yres);
      }
      __syncthreads(); 

      j += last; //blockDim.x;
      if (j > local_chunk_length) last = local_chunk_length%last; //blockDim.x;
      for (int i=0; i<last; i++)
      {
         int reg = (maxx[i]-minx[i])*(maxy[i]-miny[i]);
         // render pixel threadIdx.x of particle i
         if (threadIdx.x < reg)
         {
           // global pixel coordinates
           x = threadIdx.x/(maxy[i]-miny[i]) + minx[i];
           y = threadIdx.x%(maxy[i]-miny[i]) + miny[i];
           // global pixel index = x*dparams.yres+y
           // localx = x-xo,   localy = y-yo 
           int lp = (x-xo)*(tile_sidey+2*width) + y-yo;  //local pixel index
       //    if (lp >= tileBsize) printf("lp = %d, tile=%d, x=%d, y=%d xr =%f \n",lp,tile,x,y,posx[i]);
           float dsq = (y-posy[i])*(y-posy[i]) + (x-posx[i])*(x-posx[i]);
           if (dsq<radsq[i])
           {
             float att = __expf(stp[i]*dsq);
             Btile[lp].r += -att*e[i].r;
             Btile[lp].g += -att*e[i].g;
             Btile[lp].b += -att*e[i].b;
           }
           else
           {
             Btile[lp].r += 0.0f;
             Btile[lp].g += 0.0f;
             Btile[lp].b += 0.0f;
           }
          }
      }
      __syncthreads();  
  }

  //update inner tile in the global image
  int k0 = width*(tile_sidey+2*width) + width; // starting point
  for (int i=threadIdx.x; i<tile_sidex*tile_sidey; i=i+blockDim.x) 
  {
     j = k0 + i + (i/tile_sidey)*2*width; //add correction due to the boundary
     k = pixelLocalToGlobal(j,xo,yo,width,tile_sidey);
     pic[k].r += Btile[j].r;
     pic[k].g += Btile[j].g;
     pic[k].b += Btile[j].b;
  }
  __syncthreads();

// update boundary in 3 steps: 
// 1. columns

  int ymax = yo + tile_sidey+2*width;
  int xmax = xo + tile_sidex+2*width;
  int step = blockDim.x/2;

  if ((threadIdx.x < step)  && (yo > 0))
  {
    k0 = width*(tile_sidey+2*width);
    for (int i = threadIdx.x; i<tile_sidex*width; i=i+step) 
    {
      j = k0 + i + (i/width)*(tile_sidey+width); //add correction due to the boundary
      k = pixelLocalToGlobal(j,xo,yo,width,tile_sidey);
      pic1[k].r += Btile[j].r;
      pic1[k].g += Btile[j].g;
      pic1[k].b += Btile[j].b;
    }
  }
  else if ((threadIdx.x >= step)  && (ymax < dparams.yres))
  {
    k0 = width*(tile_sidey+2*width) + width + tile_sidey; 
    for (int i = threadIdx.x - step; i<tile_sidex*width; i=i+step) 
    {
      j = k0 + i + (i/width)*(tile_sidey+width); //add correction due to the boundary
      k = pixelLocalToGlobal(j,xo,yo,width,tile_sidey);
      pic1[k].r += Btile[j].r;
      pic1[k].g += Btile[j].g;
      pic1[k].b += Btile[j].b;
    }
  }
  __syncthreads();

// 2. rows
  if ((threadIdx.x < step) && (xo > 0))
  {
    k0 = width; 
    for (int i=threadIdx.x; i<tile_sidey*width; i=i+step) 
    {
      j = k0 + i + (i/tile_sidey)*2*width; //add correction due to the boundary
      k = pixelLocalToGlobal(j,xo,yo,width,tile_sidey);
      pic2[k].r += Btile[j].r;
      pic2[k].g += Btile[j].g;
      pic2[k].b += Btile[j].b;
    }
  }
  else if ((threadIdx.x >= step)  && (xmax < dparams.xres))
  {
    k0 = width + (width+tile_sidex)*(tile_sidey+2*width); // starting point
    for (int i=threadIdx.x - step; i<tile_sidey*width; i=i+step) 
    {
      j = k0 + i + (i/tile_sidey)*2*width; //add correction due to the boundary
      k = pixelLocalToGlobal(j,xo,yo,width,tile_sidey);
      pic2[k].r += Btile[j].r;
      pic2[k].g += Btile[j].g;
      pic2[k].b += Btile[j].b;
    }
  }
  __syncthreads();

// 3. corners
// dimension corners = 1/4 dimension blocks
  int i;
  if ((threadIdx.x < blockDim.x/4) && (xo > 0) && (yo > 0))
  {
     j = threadIdx.x + (threadIdx.x/width)*(tile_sidey+width);
     k = pixelLocalToGlobal(j,xo,yo,width,tile_sidey);
     pic3[k].r += Btile[j].r;
     pic3[k].g += Btile[j].g;
     pic3[k].b += Btile[j].b;
  }
  else if ((threadIdx.x >= blockDim.x/4 && threadIdx.x < blockDim.x/2) && (xo > 0) && (ymax < dparams.yres))
  {
     k0 = width + tile_sidey; 
     i = threadIdx.x - blockDim.x/4; 
     j = k0 + i + (i/width)*(tile_sidey+width);
     k = pixelLocalToGlobal(j,xo,yo,width,tile_sidey);
     pic3[k].r += Btile[j].r;
     pic3[k].g += Btile[j].g;
     pic3[k].b += Btile[j].b;
  }
  else if ((threadIdx.x >= blockDim.x/2 && threadIdx.x < 3*blockDim.x/4) && (xmax < dparams.xres) && (yo > 0))
  {
     k0 = (width + tile_sidex)*(tile_sidey+2*width);
     i = threadIdx.x - blockDim.x/2; 
     j = k0 + i + (i/width)*(tile_sidey+width);
     k = pixelLocalToGlobal(j,xo,yo,width,tile_sidey);
     pic3[k].r += Btile[j].r;
     pic3[k].g += Btile[j].g;
     pic3[k].b += Btile[j].b;
  }
  else if ((threadIdx.x >= 3*blockDim.x/4) && (xmax < dparams.xres) && (ymax < dparams.yres))
  {
     k0 = (width + tile_sidex)*(tile_sidey+2*width) + width + tile_sidey;
     i = threadIdx.x - 3*blockDim.x/4; 
     j = k0 + i + (i/width)*(tile_sidey+width);
     k = pixelLocalToGlobal(j,xo,yo,width,tile_sidey);
     pic3[k].r += Btile[j].r;
     pic3[k].g += Btile[j].g;
     pic3[k].b += Btile[j].b;
  }

}


//device render function k_renderC3
// each thread render a particle
__global__ void k_renderC3
  (int n, cu_particle_sim *part, int *index)
{
   //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >= n) return;
 
  cu_particle_sim p = part[m];
  float rfacr = dparams.rfac*p.r;
  int x = int(p.x-rfacr+1.f);
  x = max(x,0);
  int y = int(p.y-rfacr+1.f);
  y = max(y,0);
  index[m] = x*dparams.yres+y; 
  //pixel = -p.e
}

__global__ void k_add_images(int n, cu_color *pic, cu_color *pic1, cu_color *pic2, cu_color *pic3)
{
   //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >n) return;

   pic[m].r += pic1[m].r + pic2[m].r + pic3[m].r;
   pic[m].g += pic1[m].g + pic2[m].g + pic3[m].g;
   pic[m].b += pic1[m].b + pic2[m].b + pic3[m].b;
}

__global__ void k_addC3(int nC3, int *index, cu_particle_sim *part, cu_color *pic)
{
   //first get the index m of this thread
  int m=blockIdx.x *blockDim.x + threadIdx.x;
  if (m >= nC3) return;

  pic[index[m]].r += - part[m].e.r;
  pic[index[m]].g += - part[m].e.g;
  pic[index[m]].b += - part[m].e.b;
  
}

// check for non-active and big particles to remove from the device
struct particle_notValid
  {
    __host__ __device__
    bool operator()(const int flag)
    {
      return (flag < 0);
    }
  };

// check for active big particles to copy back to the host
struct reg_notValid
  {
    __host__ __device__
    bool operator()(const int flag)
    {
      return (flag==-2);
    }
  };

struct sum_op
{
  __host__ __device__
  cu_particle_sim operator()(cu_particle_sim& p1, cu_particle_sim& p2) const{

    cu_particle_sim sum;
    sum = p1;
    sum.e.r = p1.e.r + p2.e.r;
    sum.e.g = p1.e.g + p2.e.g;
    sum.e.b = p1.e.b + p2.e.b;

    return sum; 
   } 
};

#endif

