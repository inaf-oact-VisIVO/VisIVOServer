#include "hip/hip_runtime.h"
/*
Try accelerating splotch with CUDA. July 2009.
Copyright things go here.
*/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <cutil_inline.h>

#include "cxxsupport/lsconstants.h"
#include "cxxsupport/string_utils.h"
#include "splotch/splotchutils.h"
#include "kernel/transform.h"

#include "cuda/splotch_kernel.cu"
#include "cuda/splotch_cuda.h"
#include "cuda/CuPolicy.h"


using namespace std;

template<typename T> T findParamWithoutChange
  (paramfile *param, std::string &key, T &deflt)
  {
  return param->param_present(key) ? param->find<T>(key) : deflt;
  }

#define CLEAR_MEM(p) if(p) {cutilSafeCall(hipFree(p)); p=0;}


void getCuTransformParams(cu_param &para_trans,
paramfile &params, vec3 &campos, vec3 &lookat, vec3 &sky)
  {
  int xres = params.find<int>("xres",800),
      yres = params.find<int>("yres",xres);
  double fov = params.find<double>("fov",45); //in degrees
  double fovfct = tan(fov*0.5*degr2rad);
  float64 xfac=0.0, dist=0.0;

  sky.Normalize();
  vec3 zaxis = (lookat-campos).Norm();
  vec3 xaxis = crossprod (sky,zaxis).Norm();
  vec3 yaxis = crossprod (zaxis,xaxis);
  TRANSFORM trans;
  trans.Make_General_Transform
        (TRANSMAT(xaxis.x,xaxis.y,xaxis.z,
                  yaxis.x,yaxis.y,yaxis.z,
                  zaxis.x,zaxis.y,zaxis.z,
                  0,0,0));
  trans.Invert();
  TRANSFORM trans2;
  trans2.Make_Translation_Transform(-campos);
  trans2.Add_Transform(trans);
  trans=trans2;
  bool projection = params.find<bool>("projection",true);

  if (!projection)
    {
    float64 dist= (campos-lookat).Length();
    float64 xfac=1./(fovfct*dist);
    cout << " Field of fiew: " << 1./xfac*2. << endl;
    }

  float minrad_pix = params.find<float>("minrad_pix",1.);

  //retrieve the parameters for transformation
  for (int i=0; i<12; i++)
    para_trans.p[i] =trans.Matrix().p[i];
  para_trans.projection=projection;
  para_trans.xres=xres;
  para_trans.yres=yres;
  para_trans.fovfct=fovfct;
  para_trans.dist=dist;
  para_trans.xfac=xfac;
  para_trans.minrad_pix=minrad_pix;
  }


void cu_init(int devID, int nP, cu_gpu_vars* pgv, paramfile &fparams, vec3 &campos, vec3 &lookat, vec3 &sky)
  {
  hipSetDevice (devID); // initialize cuda runtime
  
  //allocate device memory for particle data
  size_t s = pgv->policy->GetSizeDPD(nP);
  //one more space allocated for the dumb
  cutilSafeCall(hipMalloc((void**) &pgv->d_pd, s +sizeof(cu_particle_sim)));
  
  //now prepare memory for d_particle_splotch.
  //one more for dums
  s = nP* sizeof(cu_particle_splotch);
  cutilSafeCall( hipMalloc((void**) &pgv->d_ps_render, s+sizeof(cu_particle_splotch)));

  size_t size = pgv->policy->GetFBufSize() <<20;
  cutilSafeCall( hipMalloc((void**) &pgv->d_fbuf, size)); 

  //retrieve parameters
  cu_param tparams;
  getCuTransformParams(tparams,fparams,campos,lookat,sky);

  tparams.zmaxval   = fparams.find<float>("zmax",1.e23);
  tparams.zminval   = fparams.find<float>("zmin",0.0);
  tparams.ptypes    = fparams.find<int>("ptypes",1);

  for(int itype=0; itype<tparams.ptypes; itype++)
    {
    tparams.brightness[itype] = fparams.find<double>("brightness"+dataToString(itype),1.);
    tparams.col_vector[itype] = fparams.find<bool>("color_is_vector"+dataToString(itype),false);
    }
  tparams.rfac=1.5;

  //dump parameters to device
  cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dparams), &tparams, sizeof(cu_param) ));
  }

void cu_allocate_particles(unsigned int nP, cu_gpu_vars* pgv)
  {
  //now resize d_particle_splotch.
  //one more for dums
  size_t s = (nP+1)* sizeof(cu_particle_splotch);
  cutilSafeCall( hipMalloc((void**) &pgv->d_ps_render, s)); 
  }

void cu_copy_particles_to_device(cu_particle_sim* h_pd, unsigned int n, cu_gpu_vars* pgv)
  {
  //copy particle data to device
  size_t s = pgv->policy->GetSizeDPD(n);
  cutilSafeCall(hipMemcpy(pgv->d_pd, h_pd, s, hipMemcpyHostToDevice) );
  }


void cu_transform (unsigned int n, cu_particle_splotch *h_ps, cu_gpu_vars* pgv)
  {

  //Get block dim and grid dim from pgv->policy object
  dim3 dimGrid, dimBlock;
  pgv->policy->GetDimsBlockGrid(n, &dimGrid, &dimBlock);

  //call device transformation
  k_transform<<<dimGrid,dimBlock>>>(pgv->d_pd, pgv->d_ps_render, n);

  //copy the result out
  size_t size = n* sizeof(cu_particle_splotch);
  cutilSafeCall(hipMemcpy(h_ps, pgv->d_ps_render, size, hipMemcpyDeviceToHost) );

  }

void cu_init_colormap(cu_colormap_info h_info, cu_gpu_vars* pgv)
  {
  //allocate memories for colormap and ptype_points and dump host data into it
  size_t size =sizeof(cu_color_map_entry)*h_info.mapSize;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dmap), h_info.map, size) );
  //type
  size =sizeof(int)*h_info.ptypes;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ptype_points), h_info.ptype_points, size) );

  //set fields of global variable pgv->d_colormap_info
  pgv->colormap_size   = h_info.mapSize;
  pgv->colormap_ptypes = h_info.ptypes;
  }

void cu_colorize(int n, cu_gpu_vars* pgv)
  {

  //fetch grid dim and block dim and call device
  dim3 dimGrid, dimBlock;
  pgv->policy->GetDimsBlockGrid(n, &dimGrid, &dimBlock);

  hipEvent_t start,stop;
  cutilSafeCall( hipEventCreate(&start));
  cutilSafeCall( hipEventCreate(&stop));
  cutilSafeCall( hipEventRecord( start, 0));

  k_colorize<<<dimGrid,dimBlock>>>(n, pgv->d_ps_render, pgv->colormap_size, pgv->colormap_ptypes);

  cutilSafeCall( hipEventRecord( stop, 0));
  cutilSafeCall( hipEventSynchronize(stop));
 // float elapsedTime;
 // cutilSafeCall( hipEventElapsedTime(&elapsedTime,start,stop));
  cutilSafeCall( hipEventDestroy(start));
  cutilSafeCall( hipEventDestroy(stop));

  //particle_splotch memory on device will be freed in cu_end
  }


void cu_copy_particles_to_render(cu_particle_splotch *p,
  int n, cu_gpu_vars* pgv)
  {
  //copy filtered particles into device
  size_t size = n *sizeof(cu_particle_splotch);
  cutilSafeCall(hipMemcpy(pgv->d_ps_render, p,size,
    hipMemcpyHostToDevice) );
  }

void cu_render1
  (int nP, bool a_eq_e, float grayabsorb, cu_gpu_vars* pgv)
  {
  hipEvent_t start,stop;
  cutilSafeCall( hipEventCreate(&start));
  cutilSafeCall( hipEventCreate(&stop));
  cutilSafeCall( hipEventRecord( start, 0));
 
  //get dims from pgv->policy object first
  dim3 dimGrid, dimBlock;
  pgv->policy->GetDimsBlockGrid(nP, &dimGrid, &dimBlock);

  //call device
  k_render1<<<dimGrid, dimBlock>>>(pgv->d_ps_render, nP,
    pgv->d_fbuf, a_eq_e, grayabsorb, pgv->colormap_size, pgv->colormap_ptypes);

  cutilSafeCall( hipEventRecord( stop, 0));
  cutilSafeCall( hipEventSynchronize(stop));
//  float elapsedTime;
//  cutilSafeCall( hipEventElapsedTime(&elapsedTime,start,stop));
  cutilSafeCall( hipEventDestroy(start));
  cutilSafeCall( hipEventDestroy(stop));
  }


void cu_get_fbuf
  (void *h_fbuf, bool a_eq_e, unsigned long n, cu_gpu_vars* pgv)
  {
  size_t size;
  if (a_eq_e)
    size =n* sizeof(cu_fragment_AeqE);
  else
    size =n* sizeof(cu_fragment_AneqE);

  cutilSafeCall( hipMemcpy(h_fbuf, pgv->d_fbuf,size,
    hipMemcpyDeviceToHost)) ;
  }

void cu_end(cu_gpu_vars* pgv)
  {
  CLEAR_MEM((pgv->d_pd));
  CLEAR_MEM((pgv->d_ps_render));
  CLEAR_MEM((pgv->d_fbuf));

  hipDeviceReset();

  delete pgv->policy;
  }

int cu_get_chunk_particle_count(paramfile &params, CuPolicy* policy, size_t psize, float pfactor)
  {
   int gMemSize = policy->GetGMemSize();
   int fBufSize = policy->GetFBufSize();
   if (gMemSize <= fBufSize) return 0;

  // float factor =params.find<float>("particle_mem_factor", 3);
   int spareMem = 10;
   int arrayParticleSize = gMemSize - fBufSize - spareMem;

   return (int) (arrayParticleSize/psize/pfactor)*(1<<20);
  }
